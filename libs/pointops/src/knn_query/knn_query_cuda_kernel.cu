#include "hip/hip_runtime.h"
#include "../cuda_utils.h"
#include "knn_query_cuda_kernel.h"
#include <hip/hip_fp16.h>

namespace knn_query_utils {

template <typename DType>
__device__ void swap(DType* x, DType* y) {
    DType tmp = *x;
    *x = *y;
    *y = tmp;
}

__device__ void reheap(float* dist, int* idx, int k) {
    int root = 0;
    int child = root * 2 + 1;
    while (child < k) {
        if (child + 1 < k && dist[child + 1] > dist[child])
            child++;
        if (dist[root] > dist[child])
            return;
        swap<float>(&dist[root], &dist[child]);
        swap<int>(&idx[root], &idx[child]);
        root = child;
        child = root * 2 + 1;
    }
}

__device__ void heap_sort(float* dist, int* idx, int k) {
    for (int i = k - 1; i > 0; i--) {
        swap<float>(&dist[0], &dist[i]);
        swap<int>(&idx[0], &idx[i]);
        reheap(dist, idx, i);
    }
}

__device__ int get_bt_idx(int idx, const int* offset) {
    int i = 0;
    while (true) {
        if (idx < offset[i])
            break;
        i++;
    }
    return i;
}

template <typename scalar_t>
__device__ float load_as_float(const scalar_t* ptr) {
    return static_cast<float>(*ptr);
}

template <>
__device__ float load_as_float<at::Half>(const at::Half* ptr) {
    return __half2float(*reinterpret_cast<const __half*>(ptr));
}

}  // namespace knn_query_utils

template <typename scalar_t>
__global__ void knn_query_cuda_kernel_template(
    int m, int nsample,
    const scalar_t* __restrict__ xyz,
    const scalar_t* __restrict__ new_xyz,
    const int* __restrict__ offset,
    const int* __restrict__ new_offset,
    int* __restrict__ idx,
    scalar_t* __restrict__ dist2_out) {

    int pt_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (pt_idx >= m) return;

    const scalar_t* this_query = new_xyz + pt_idx * 3;
    int* this_idx = idx + pt_idx * nsample;
    scalar_t* this_dist2 = dist2_out + pt_idx * nsample;

    int bt_idx = knn_query_utils::get_bt_idx(pt_idx, new_offset);
    int start = (bt_idx == 0) ? 0 : offset[bt_idx - 1];
    int end = offset[bt_idx];

    float new_x = knn_query_utils::load_as_float(this_query + 0);
    float new_y = knn_query_utils::load_as_float(this_query + 1);
    float new_z = knn_query_utils::load_as_float(this_query + 2);

    float best_dist[128];
    int best_idx[128];
    for (int i = 0; i < nsample; i++) {
        best_dist[i] = 1e10f;
        best_idx[i] = -1;
    }

    for (int i = start; i < end; i++) {
        float x = knn_query_utils::load_as_float(xyz + i * 3 + 0);
        float y = knn_query_utils::load_as_float(xyz + i * 3 + 1);
        float z = knn_query_utils::load_as_float(xyz + i * 3 + 2);
        float d2 = (new_x - x) * (new_x - x)
                 + (new_y - y) * (new_y - y)
                 + (new_z - z) * (new_z - z);
        if (d2 < best_dist[0]) {
            best_dist[0] = d2;
            best_idx[0] = i;
            knn_query_utils::reheap(best_dist, best_idx, nsample);
        }
    }

    knn_query_utils::heap_sort(best_dist, best_idx, nsample);

    for (int i = 0; i < nsample; i++) {
        this_idx[i] = best_idx[i];
        this_dist2[i] = static_cast<scalar_t>(best_dist[i]);
    }
}

template <typename scalar_t>
void knn_query_cuda_launcher(
    int m, int nsample,
    const scalar_t* xyz,
    const scalar_t* new_xyz,
    const int* offset,
    const int* new_offset,
    int* idx,
    scalar_t* dist2) {

    dim3 blocks(DIVUP(m, THREADS_PER_BLOCK));
    dim3 threads(THREADS_PER_BLOCK);
    knn_query_cuda_kernel_template<scalar_t><<<blocks, threads, 0>>>(
        m, nsample, xyz, new_xyz, offset, new_offset, idx, dist2
    );
}

// Explicit instantiation
template void knn_query_cuda_launcher<float>(int, int, const float*, const float*, const int*, const int*, int*, float*);
template void knn_query_cuda_launcher<at::Half>(int, int, const at::Half*, const at::Half*, const int*, const int*, int*, at::Half*);